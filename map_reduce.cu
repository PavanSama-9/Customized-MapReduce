#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "config.cuh"

using namespace std;

extern __device__ void mapper(input_type *input, KeyValuePair *pairs);
extern __device__ void reducer(KeyValuePair *pairs, int len, output_type *output);

/*
 * Macro to check for GPU errors
 */
#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(
    hipError_t code,
    const char *file,
    int line,
    bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n",
            hipGetErrorString(code), file, line);
        exit(code);
    }
}

/*
 * An operator definition that allows comparisons between two KeyValuePairs
 * This does a byte by byte comparison of the key, and returns True if
 * the first pair has a key less than second pair.
 */
struct keyValueCompare {
    __host__ __device__ bool operator() (const KeyValuePair lhs, const KeyValuePair rhs) {
        char a=lhs.value[0];
        char b=rhs.value[0];
        return a<b;
    }
};

// Declare mapper and reducer functions
void cudaMap(input_type *input, KeyValuePair *pairs);
void cudaReduce(KeyValuePair *pairs, output_type *output);

/*
 * Mapping Kernel: Since each mapper runs independently of each other, we can
 * give each thread its own input to process and a disjoint space where it can`
 * store the key/value pairs it produces.
 */
__global__ void mapKernel(input_type *input, KeyValuePair *pairs) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x;
        i < NUM_INPUT;
        i += blockDim.x * gridDim.x) {
        mapper(&input[i], &pairs[i * NUM_KEYS]);
    }
}

/*
 * Reducing Kernel: Given a sorted array of keys, find the range corresponding
 * to each thread and run the reducer on that set of key/value pairs.
 */
__global__ void reduceKernel(KeyValuePair *pairs, output_type *output) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x;
        i < NUM_OUTPUT;
        i += blockDim.x * gridDim.x) {
        int startIndex = 0;
        int count = 0;
        int valueSize = 0;
        int j;

        for (j = 1; j < NUM_INPUT * NUM_KEYS; j++) {
            if (keyValueCompare()(pairs[j - 1], pairs[j])) {
                if (count == i) {
                    // This thread has found the correct number
                    // There is a bit of warp divergence here as some threads
                    // break before others, but we still make the most out of it
                    // by calling the reducer at the very end, so there is not
                    // any warp divergence where the bulk of the computation
                    // should occur (the reducer).
                    break;
                } else {
                    count++;
                    startIndex = j;
                }
            }
        }

        if (count < i) {
            // This thread doesn't need to process a key. We won't get here, but
            // this code is just there for assurance.
            return;
        }

        valueSize = j+startIndex;
        
        // Run the reducer
        reducer(pairs , valueSize, output);
    }
}

/*
 * The main function that runs the bulk of the MapReduce job. Space is allocated
 * on the GPU, inputs are copied. The mapper is run. The key/value pairs are
 * sorted. The reducer is run. Output data is copied back from the GPU and
 * returned.
 */
void runMapReduce(input_type *input, output_type *output) {
    // Create device pointers
    input_type   *dev_input;
    output_type  *dev_output;
    KeyValuePair *dev_pairs;

    // Determine sizes in bytes
    size_t input_size = NUM_INPUT * sizeof(input_type);
    size_t output_size = NUM_OUTPUT * sizeof(output_type);
    size_t pairs_size = NUM_INPUT * NUM_KEYS * sizeof(KeyValuePair);

    // Initialize device memory (we can utilize more space by waiting to
    // initialize the output array until we're done with the input array)
    hipMalloc(&dev_input, input_size);
    hipMalloc(&dev_pairs, pairs_size);

    // Copy input data over
    hipMemcpy(dev_input, input, input_size, hipMemcpyHostToDevice);
    //hipMemset(dev_pairs, 0, pairs_size);

    // Run the mapper kernel
    cudaMap(dev_input, dev_pairs);

    // Convert the pointer to device memory for the key/value pairs that is
    // recognizable by the cuda thrust library
    thrust::device_ptr<KeyValuePair> dev_ptr(dev_pairs);

    // Sort the key/value pairs. By using the thrust library, we don't have to
    // write this code ourselves, and it's already optimized for parallel
    // computation
    thrust::sort(dev_ptr, dev_ptr + NUM_INPUT * NUM_KEYS, keyValueCompare());

    // Free GPU space for the input
    hipFree(dev_input);
    // Allocate GPU space for the output
    hipMalloc(&dev_output, output_size);

    // Run the reducer kernel
    cudaReduce(dev_pairs, dev_output);

    // Allocate space on the host for the output array and copy the data to it
    hipMemcpy(output, dev_output, output_size, hipMemcpyDeviceToHost);

    // Free GPU memory for the key/value pairs and output array
    hipFree(dev_pairs);
    hipFree(dev_output);
}

/*
 * Function to call the cuda map kernel and ensure no errors occur
 */
void cudaMap(input_type *input, KeyValuePair *pairs) {
    mapKernel<<<GRID_SIZE, BLOCK_SIZE>>>(input, pairs);
    gpuErrChk( hipPeekAtLastError() );
    gpuErrChk( hipDeviceSynchronize() );
}

/*
 * Function to call the cuda reduce kernel and ensure no errors occur
 */
void cudaReduce(KeyValuePair *pairs, output_type *output) {
    reduceKernel<<<GRID_SIZE, BLOCK_SIZE>>>(pairs, output);
    gpuErrChk( hipPeekAtLastError() );
    gpuErrChk( hipDeviceSynchronize() );
}