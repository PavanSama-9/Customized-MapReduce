#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include<iostream>
#include "config.cuh"
#include<map>
#include <sstream>
#include <vector>
#include <algorithm> 
#include <cassert>
#define maxWordSize 1024
using namespace std;
vector < string > v1;

/*
 * Mapping function to be run for each input. The input must be read from memory
 * and the the key/value output must be stored in memory at pairs. Multiple
 * pairs may be stored at the next postiion in pairs, but the maximum number of
 * key/value pairs stored must not exceed NUM_KEYS.
 */
__device__ void mapper(input_type *input, KeyValuePair *pairs) {
     pairs->key =0;	
     for(int i=0;input->inputName[i]!='\0';i++)
     {
     	 
     	pairs->value[i] =input->inputName[i];

     }
}

/*
 * Reducing function to be run for each set of key/value pairs that share the
 * same key. len key/value pairs may be read from memory, and the output
 * generated from these pairs must be stored at output in memory.
 */
__device__ void reducer(KeyValuePair *pairs, int len,output_type *output) {

	for(int k=0;k<(len-1);k++)
	{
		int wordCount=0;
		int stringCount=1;
		int size=0;
        int duplicatWordCount=0;
        int duplicateCount=1;

		for(int w=0;((pairs+k)->value[w])!='\0';w++)
		{
			size++;
		}
		for(int l=k+1;l<len;l++)
		{
			wordCount=0;
			for(int m=0;m<size;m++)
			{
				if((((pairs+k)->value[m])==((pairs+l)->value[m])))
				{
					if(((pairs+l)->value[size])!='\0')
					{
                            break;
					}
					else
					{
						wordCount++;
					}
					
				}
				if(size==wordCount)
				{
					stringCount++;
				}
			}
		}
        for(int v=k-1;v>=0;v--)
        {
            duplicatWordCount=0;
            for(int m=0;m<size;m++)
            {
                if((((pairs+k)->value[m])==((pairs+v)->value[m])))
                {
                    if(((pairs+v)->value[size])!='\0')
                    {
                            
                    }
                    else
                    {
                        duplicatWordCount++;
                    }
                    
                }
                if(size==duplicatWordCount)
                {
                    duplicateCount++;
                }
            }

        }
            if(duplicateCount==1)
            {
                    (output+k)->x = stringCount;
                    for(int i=0;((pairs+k)->value[i])!='\0';i++)
                    {
                         (output+k)->y[i] = (pairs+k)->value[i] ;
                    } 
            }
            else
            {
                (output+k)->x = 0;
                for(int i=0;((pairs+k)->value[i])!='\0';i++)
                    {
                         (output+k)->y[i] ='\0';
                    } 
            }

	}
 }

void StringWithoutSigns(char *sign)
{
	int len=strlen(sign);
	if(sign[len-1]>0 && sign[len-1]>32 && sign[len-1]<65)
	{
		sign[len-1]=0;
		StringWithoutSigns(sign);
	}
}
void read_words (FILE *f, map<string, int> &m) 
{
    char x[maxWordSize];
    cout<<"Input Data"<<endl;	
    cout<<"*************************"<<endl;
    while (fscanf(f, " %1023s", x) == 1) 
	{
		StringWithoutSigns(x);
		m[x]++;
		string s = std::string(x);
		v1.push_back( s );
     cout<<s<<endl;
    }
    cout<<"*************************"<<endl;
}

/*
 * Main function that runs a map reduce job.
 */
int main(int argc, char const *argv[]) {
    // Allocate host memory
    size_t input_size = NUM_INPUT * sizeof(input_type);
    size_t output_size = NUM_OUTPUT * sizeof(output_type);
    input_type *input = (input_type *) malloc(input_size);
    output_type *output = (output_type *) malloc(output_size);
    map<string, int> m;
	FILE *inputFile;
    hipEvent_t start, stop;
    float milliseconds = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	inputFile = fopen("/home/pavankum.sama/fileInput.txt","r");
	read_words(inputFile, m);


  //  printf("Generating %d Test Points\n", NUM_INPUT);
    for (size_t i = 0; i < NUM_INPUT; i++) {
     
         	string s =v1[i]; 
    	    int n = s.length(); 
          // declaring character array 
          char char_array[n + 1]; 
  
         // copying the contents of the 
        // string to char array 
         strcpy(char_array, s.c_str()); 
	    for (int j = 0; j < n; j++) 
	    {
	    	input[i].inputName[j]=char_array[j];
	    }
    }

   
    hipEventRecord(start);
     // Run the Map Reduce Job
    runMapReduce(input, output);
    hipEventRecord(stop);

    // Iterate through the output array
    cout<<"Map Reduce wordCount"<<endl;
    cout<<"*************************"<<endl;
    for (size_t i = 0; i <NUM_OUTPUT; i++) {
        if(output[i].x!=0)
        cout<<"("<<output[i].y<<","<<output[i].x<<")"<<endl;
    }
    cout<<"*************************"<<endl;

    hipEventElapsedTime(&milliseconds, start, stop);
    cout<<"Kernel Exceuction completed"<<endl; 
    cout<<"GPU Exceution Time is: "<<milliseconds<<" ms"<<endl; 

    // Free host memory
    free(input);
    free(output);

    return 0;
}